
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

// const size_t Nx = 64;
// const size_t Ny = 64;
// const size_t Nz = 64;
size_t Nx = 128;
size_t Ny = 512;
size_t Nz = 512;

const size_t nrepeat = 100;

__host__ __device__ __forceinline__ int index(const int i, const int j,
                                              const int k, const dim3 sizes) {
    const int istride = 1;
    const int jstride = sizes.x;
    const int kstride = sizes.x * sizes.y;
    return i * istride + j * jstride + k * kstride;
}
__host__ __device__ __forceinline__ int
index_strides(const int i, const int j, const int k, const dim3 strides) {
    return i * strides.x + j * strides.y + k * strides.z;
}

// TODO parameterize ldg
//__global__ void laplace3d(double *d, double *n) {
//    int i = threadIdx.x + blockIdx.x * blockDim.x;
//    int j = threadIdx.y + blockIdx.y * blockDim.y;
//    int k = threadIdx.z + blockIdx.z * blockDim.z;
//
//    if (i > 0 && i < Nx - 1)
//        if (j > 0 && j < Ny - 1)
//            if (k > 0 && k < Nz - 1)
//                d[index(i, j, k)] =
//                    1. / 2. * ( //
//                                  __ldg(&n[index(i - 1, j, k)]) +
//                                  __ldg(&n[index(i + 1, j, k)]) //
//                                  + __ldg(&n[index(i, j - 1, k)]) +
//                                  __ldg(&n[index(i, j + 1, k)]) //
//                                  + __ldg(&n[index(i, j, k - 1)]) +
//                                  __ldg(&n[index(i, j, k + 1)]) //
//                                  - 6. * __ldg(&n[index(i, j, k)]));
//}

__global__ void laplace3d_strides(double *d, double *n, const dim3 sizes,
                                  const dim3 strides) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i > 0 && i < sizes.x - 1)
        if (j > 0 && j < sizes.y - 1)
            if (k > 0 && k < sizes.z - 1)
                d[index_strides(i, j, k, strides)] =
                    1. / 2. *
                    (                                                    //
                        __ldg(&n[index_strides(i - 1, j, k, strides)])   //
                        + __ldg(&n[index_strides(i + 1, j, k, strides)]) //
                        + __ldg(&n[index_strides(i, j - 1, k, strides)]) //
                        + __ldg(&n[index_strides(i, j + 1, k, strides)]) //
                        + __ldg(&n[index_strides(i, j, k - 1, strides)]) //
                        + __ldg(&n[index_strides(i, j, k + 1, strides)]) //
                        - 6. * __ldg(&n[index_strides(i, j, k, strides)]));
}

__host__ __device__ __forceinline__ int index_smem(const int i, const int j,
                                                   const int k) {
    return (i + 1) + (j + 1) * (blockDim.x + 2) +
           (k + 1) * (blockDim.x + 2) * (blockDim.y + 2);
}

__global__ void laplace3d_smem(double *d, double *n, const dim3 sizes,
                               const dim3 strides) {
    extern __shared__ double smem[];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    int ii = threadIdx.x;
    int jj = threadIdx.y;
    int kk = threadIdx.z;

    smem[index_smem(ii, jj, kk)] = __ldg(&n[index_strides(i, j, k, strides)]);
    if (ii == 0) {
        if (jj != 0 && kk != 0 && //
            jj != blockDim.y - 1 && kk != blockDim.z - 1) {
            if (i > 0)
                smem[index_smem(-1, jj, kk)] =
                    __ldg(&n[index_strides(i - 1, j, k, strides)]);
            if (i < sizes.x - 1)
                smem[index_smem(blockDim.x, jj, kk)] =
                    __ldg(&n[index_strides(i + 1, j, k, strides)]);
        }
    }
    if (jj == 0) {
        if (ii != 0 && kk != 0 && //
            ii != blockDim.x - 1 && kk != blockDim.z - 1) {
            if (j > 0)
                smem[index_smem(ii, -1, kk)] =
                    __ldg(&n[index_strides(i, j - 1, k, strides)]);
            if (j < sizes.y - 1)
                smem[index_smem(ii, blockDim.y, kk)] =
                    __ldg(&n[index_strides(i, j + 1, k, strides)]);
        }
    }
    if (kk == 0) {
        if (ii != 0 && jj != 0 && //
            ii != blockDim.x - 1 && jj != blockDim.y - 1) {
            if (k > 0)
                smem[index_smem(ii, jj, -1)] =
                    __ldg(&n[index_strides(i, j, k - 1, strides)]);
            if (k < sizes.z - 1)
                smem[index_smem(ii, jj, blockDim.z)] =
                    __ldg(&n[index_strides(i, j, k + 1, strides)]);
        }
    }

    if (i > 0 && i < sizes.x - 1)
        if (j > 0 && j < sizes.y - 1)
            if (k > 0 && k < sizes.z - 1)
                d[index_strides(i, j, k, strides)] =
                    1. / 2. * (                                      //
                                  smem[index_smem(ii - 1, jj, kk)]   //
                                  + smem[index_smem(ii + 1, jj, kk)] //
                                  + smem[index_smem(ii, jj - 1, kk)] //
                                  + smem[index_smem(ii, jj + 1, kk)] //
                                  + smem[index_smem(ii, jj, kk - 1)] //
                                  + smem[index_smem(ii, jj, kk + 1)] //
                                  - 6. * smem[index_smem(ii, jj, kk)]);

    //    d[index_strides(i, j, k, strides)] = smem[index_smem(ii, jj, kk)];
}

void init(double *n, const dim3 sizes) {
    for (size_t i = 0; i < sizes.x; ++i)
        for (size_t j = 0; j < sizes.y; ++j)
            for (size_t k = 0; k < sizes.z; ++k) {
                n[index(i, j, k, sizes)] =
                    sin((double)i / ((double)sizes.x - 1.) * M_PI) *
                    sin((double)j / ((double)sizes.y - 1.) * M_PI) *
                    sin((double)k / ((double)sizes.z - 1.) * M_PI);
            }
}

void print(double *n, const dim3 sizes) {
    for (size_t i = 0; i < sizes.x; ++i) {
        std::cout << (double)i / (double)(sizes.x - 1) << " \t"
                  << 1. / -0.004 * n[index(i, sizes.y / 2, sizes.z / 2, sizes)]
                  << std::endl;
    }
}

float elapsed(hipEvent_t &start, hipEvent_t &stop) {
    float result;
    hipEventElapsedTime(&result, start, stop);
    return result;
}

void execute(dim3 threadsPerBlock, double *dd, double *dn) {
    const dim3 sizes(Nx, Ny, Nz);
    const dim3 strides(1, Nx, Nx * Ny);

    hipEvent_t start_;
    hipEvent_t stop_;
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    dim3 nBlocks(Nx / threadsPerBlock.x, Ny / threadsPerBlock.y,
                 Nz / threadsPerBlock.z);

    hipEventRecord(start_, 0);

    size_t smem_size = (threadsPerBlock.x + 2) * (threadsPerBlock.y + 2) *
                       (threadsPerBlock.z + 2);

    for (size_t i = 0; i < nrepeat; ++i)
        //        laplace3d_strides<<<nBlocks, threadsPerBlock>>>(dd, dn, sizes,
        //        strides);
        laplace3d_smem<<<nBlocks, threadsPerBlock,
                         smem_size * sizeof(double)>>>(dd, dn, sizes, strides);
    //        laplace3d<<<nBlocks, threadsPerBlock>>>(dd, dn);
    hipEventRecord(stop_, 0);
    hipEventSynchronize(stop_);

    std::cout << "# threads/block = (" << threadsPerBlock.x << "/"
              << threadsPerBlock.y << "/" << threadsPerBlock.z << "), \t";
    std::cout << "blocks = (" << nBlocks.x << "/" << nBlocks.y << "/"
              << nBlocks.z << "), \t";
    std::cout << "time = " << elapsed(start_, stop_) / (float)nrepeat << "ms"
              << std::endl;

    hipEventDestroy(start_);
    hipEventDestroy(stop_);
}

int main() {
    dim3 sizes(Nx, Ny, Nz);

    size_t total_size = Nx * Ny * Nz;
    double *d = new double[total_size];
    double *n = new double[total_size];

    init(n, sizes);

    double *dd;
    hipMalloc(&dd, sizeof(double) * total_size);
    double *dn;
    hipMalloc(&dn, sizeof(double) * total_size);

    hipMemcpy(dn, n, sizeof(double) * total_size, hipMemcpyHostToDevice);

    execute(dim3(32, 4, 4), dd, dn);
    execute(dim3(8, 8, 8), dd, dn);
    //    execute(dim3(16, 8, 8), dd, dn);
    //    execute(dim3(16, 16, 4), dd, dn);
    //    execute(dim3(32, 8, 4), dd, dn);
    //    execute(dim3(64, 4, 4), dd, dn);

    hipMemcpy(d, dd, sizeof(double) * total_size, hipMemcpyDeviceToHost);

    print(d, sizes);

    delete[] d;
    hipFree(dd);
    hipFree(dn);
}
