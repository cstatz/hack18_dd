
#include <hip/hip_runtime.h>

extern "C" {
__host__ __device__ __forceinline__ int
index_strides(const int i, const int j, const int k, const dim3 strides) {
    return i * strides.x + j * strides.y + k * strides.z;
}

__global__ void laplace3d_strides(double *d, double *n, int Nx, int Ny, int Nz, int istride, int jstride, int kstride) {
    dim3 sizes(Nx,Ny,Nz);
    dim3 strides(istride,jstride,kstride);
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;

    if (i > 0 && i < sizes.x - 1)
        if (j > 0 && j < sizes.y - 1)
            if (k > 0 && k < sizes.z - 1)
                d[index_strides(i, j, k, strides)] =
                    1. / 2. *
                    (                                                    //
                        __ldg(&n[index_strides(i - 1, j, k, strides)])   //
                        + __ldg(&n[index_strides(i + 1, j, k, strides)]) //
                        + __ldg(&n[index_strides(i, j - 1, k, strides)]) //
                        + __ldg(&n[index_strides(i, j + 1, k, strides)]) //
                        + __ldg(&n[index_strides(i, j, k - 1, strides)]) //
                        + __ldg(&n[index_strides(i, j, k + 1, strides)]) //
                        - 6. * __ldg(&n[index_strides(i, j, k, strides)]));
}

__global__ void set_val(double *d) {
    int index = threadIdx.x + blockIdx.x * blockDim.x + threadIdx.y + blockIdx.y * blockDim.y + threadIdx.z + blockIdx.z * blockDim.z;
    d[index] = 123.;
}

}

